#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <chrono>

#define BLOCK_SIZE 16 // Размер блока для GPU
#define PI 3.14159265359

// GPU ядро для глобальной памяти
__global__ void Conv_Glb(float* dConv, float* dS, int W, int H, int delta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + delta;
    int idy = blockIdx.y * blockDim.y + threadIdx.y + delta;

    float norm = 0.0f, cov = 0.0f;

    for (int ix = -delta; ix <= delta; ix++) {
        for (int iy = -delta; iy <= delta; iy++) {
            float K = expf(-(ix * ix + iy * iy) / (delta * delta));
            cov += K * dS[idx + ix + (idy + iy) * (W + BLOCK_SIZE)];
            norm += K;
        }
    }
    dConv[idx + idy * (W + BLOCK_SIZE)] = cov / norm;
}

// GPU ядро для текстурной памяти (Texture Object API)
__global__ void Conv_Tex(float* dConv, hipTextureObject_t texObj, int W, int H, int delta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + delta;
    int idy = blockIdx.y * blockDim.y + threadIdx.y + delta;

    float norm = 0.0f, cov = 0.0f;

    for (int ix = -delta; ix <= delta; ix++) {
        for (int iy = -delta; iy <= delta; iy++) {
            float K = expf(-(ix * ix + iy * iy) / (delta * delta));
            cov += K * tex2D<float>(texObj, idx + ix, idy + iy);
            norm += K;
        }
    }
    dConv[idx + idy * (W + BLOCK_SIZE)] = cov / norm;
}

int main() {
    int ws[] = { 1024, 5120, 10240 }, hs[] = { 2048, 5120, 10240 };

    for (int i = 0; i < 3; ++i) {

        int W = ws[i], H = hs[i];

        int delta = BLOCK_SIZE / 2, size = (W + BLOCK_SIZE) * (H + BLOCK_SIZE);
        size_t mem_size = sizeof(float) * size;

        float* hS = (float*)malloc(mem_size);
        float* hConv = (float*)malloc(mem_size);
        float* hdConv = (float*)malloc(mem_size);
        float* hdConvText = (float*)malloc(mem_size);

        for (int y = 0; y < H + BLOCK_SIZE; y++) {
            for (int x = 0; x < W + BLOCK_SIZE; x++) {
                hS[x + y * (W + BLOCK_SIZE)] = sinf(x * 2.0f * PI / (W + BLOCK_SIZE)) * sinf(y * 2.0f * PI / (H + BLOCK_SIZE));
            }
        }

        dim3 nThreads(BLOCK_SIZE, BLOCK_SIZE);
        dim3 nBlocks(W / BLOCK_SIZE, H / BLOCK_SIZE);

        // --------------------- CPU ---------------------
        auto startCPU = std::chrono::high_resolution_clock::now();
        for (int y = delta; y < H + delta; y++) {
            for (int x = delta; x < W + delta; x++) {
                float norm = 0.0f, cov = 0.0f;
                for (int iy = -delta; iy <= delta; iy++) {
                    for (int ix = -delta; ix <= delta; ix++) {
                        float K = expf(-(ix * ix + iy * iy) / (delta * delta));
                        cov += K * hS[(x + ix) + (y + iy) * (W + BLOCK_SIZE)];
                        norm += K;
                    }
                }
                hConv[x + y * (W + BLOCK_SIZE)] = cov / norm;
            }
        }
        auto endCPU = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float, std::milli> cpuTime = endCPU - startCPU;

        // --------------------- GPU ---------------------
        float* dS, * dConv, * dConvText;

        hipMalloc((void**)&dS, mem_size);
        hipMalloc((void**)&dConv, mem_size);

        hipEvent_t start, stop;
        float timerGlobal, timerTexture;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // --------------------- GPU-GLOBAL ---------------------
        hipMemcpy(dS, hS, mem_size, hipMemcpyHostToDevice);
        hipMemset(dConv, 0, mem_size);

        hipEventRecord(start,0);

        Conv_Glb << <nBlocks, nThreads >> > (dConv, dS, W, H, delta);
        hipDeviceSynchronize();

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&timerGlobal, start, stop);

        hipMemcpy(hdConv, dConv, mem_size, hipMemcpyDeviceToHost);

        // --------------------- GPU-TEXTURE ---------------------
        hipMalloc((void**)&dConvText, mem_size);
        hipMemset(dConvText, 0, mem_size);

        hipArray* cuArray;
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        hipMallocArray(&cuArray, &channelDesc, W + BLOCK_SIZE, H + BLOCK_SIZE);
        hipMemcpyToArray(cuArray, 0, 0, hS, mem_size, hipMemcpyHostToDevice);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;

        hipTextureDesc texDesc = {};
        texDesc.addressMode[0] = hipAddressModeClamp;
        texDesc.addressMode[1] = hipAddressModeClamp;
        texDesc.filterMode = hipFilterModePoint;
        texDesc.readMode = hipReadModeElementType;
        texDesc.normalizedCoords = false;


        hipTextureObject_t texObj = 0;
        hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

        hipEventRecord(start,0);

        Conv_Tex << <nBlocks, nThreads >> > (dConvText, texObj, W, H, delta);
        hipDeviceSynchronize();

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&timerTexture, start, stop);

        hipMemcpy(hdConvText, dConv, mem_size, hipMemcpyDeviceToHost);

        std::cout << "CPU time: " << cpuTime.count() << " ms\n";
        std::cout << "GPU (Global memory) time: " << timerGlobal << " ms\n";
        std::cout << "GPU (Texture memory) time: " << timerTexture << " ms\n";
        std::cout << "Speedup (Global): " << cpuTime.count() / timerGlobal << "x\n";
        std::cout << "Speedup (Texture): " << cpuTime.count() / timerTexture << "x\n";

        hipFree(dS);
        hipFree(dConv);
        hipFreeArray(cuArray);
        hipDestroyTextureObject(texObj);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        free(hS);
        free(hConv);
        free(hdConv);
    }

    return 0;
}
